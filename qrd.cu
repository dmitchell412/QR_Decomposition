
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <math.h>
using namespace std;

__device__
void matrix_print(double *mat, int nDim)
{
	for (int j = 0; j < nDim; ++j)
	{
		for (int i = 0; i < nDim; ++i)
		{
			printf(" %8.3f", mat[j + i * nDim]);
		}
		printf("\n");
	}
	printf("\n");
}

__device__
double dotprod(double *vec1, double *vec2, int nDim)
{
	double x = 0;
	for (int i = 0; i < nDim; ++i)
		x += vec1[i] * vec2[i];
	return x;
}

__device__
double* matmult(double *mat1, double *mat2, int nDim)
{
	double *x = new double[nDim * nDim];
	for (int i = 0; i < nDim * nDim; ++i)
		x[i] = 0;
	for (int k = 0; k < nDim; ++k)
		for (int j = 0; j < nDim; ++j)
			for (int i = 0; i < nDim; ++i)
				x[j + k * nDim] += mat1[j + i * nDim] * mat2[i + k * nDim];
	return x;
}

__device__
double l2norm(double *vec, int nDim)
{
	double x = 0;
	for (int i = 0; i < nDim; ++i)
		x += vec[i] * vec[i];
	x = sqrt(x);
	return x;
}

__device__
void transpose(double *mat, int nDim)
{
	double *x = new double[nDim * nDim];
	for (int i = 0; i < nDim * nDim; ++i)
		x[i] = mat[i];
	for (int j = 0; j < nDim; ++j)
		for (int i = 0; i < nDim; ++i)
			mat[i + j * nDim] = x[j + i * nDim];
	delete[] x;
}

__device__
void make_comp_mat(double *polynomial, double *companion, int nDim)
{
	for (int i = 0; i < nDim * nDim; ++i)
		companion[i] = 0;
	for (int i = 0; i < nDim; ++i)
		companion[i * nDim] = -polynomial[i 
		+ 1
		] / polynomial[0];
	//companion[0] = 0;
	for (int i = 0; i < nDim - 1; ++i)
	//for (int i = 1; i < nDim - 1; ++i)
		companion[i * nDim + i + 1] = 1;
}

__device__
void select_diag(double *vector, double *matrix, int nDim)
{
	for (int i = 0; i < nDim; ++i)
		vector[i] = matrix[i * nDim + i];
}

__device__
void pixel_mat_select_1d(
	double *a_image,
	double *a,
	int nDim_matrix,
	int i_image)
{
	int offset_1d = i_image * nDim_matrix;
	for (int i = 0; i < nDim_matrix; ++i)
		a[i] = a_image[i + offset_1d];
}

__device__
void pixel_mat_write_1d(
	double *a_image,
	double *a,
	int nDim_matrix,
	int i_image)
{
	int offset_1d = i_image * nDim_matrix;
	for (int i = 0; i < nDim_matrix; ++i)
	{
		a_image[i + offset_1d] = a[i];
	}
}

__device__
void pixel_mat_select_2d(
	double *a_image,
	double *a,
	int nDim_matrix,
	int i_image)
{
	int offset_2d = i_image * nDim_matrix * nDim_matrix;
	for (int i = 0; i < nDim_matrix * nDim_matrix; ++i)
		a[i] = a_image[i + offset_2d];
}

__device__
void pixel_mat_write_2d(
	double *Q_image,
	double *R_image,
	double *Q,
	double *R,
	int nDim_matrix,
	int i_image)
{
	int offset_2d = i_image * nDim_matrix * nDim_matrix;
	for (int i = 0; i < nDim_matrix * nDim_matrix; ++i)
	{
		Q_image[i + offset_2d] = Q[i];
		R_image[i + offset_2d] = R[i];
	}
}

__device__
void gram_schmidt(double *a, double *Q, double *R, int nDim)
{
	double *u = new double[nDim];
	double *v = new double[nDim];
	double l2 = 0;
	for (int i = 0; i < nDim * nDim; ++i)
		Q[i] = R[i] = 0;
	for (int i = 0; i < nDim; ++i)
		u[i] = v[i] = 0;

	for (int k = 0; k < nDim; ++k)
	{
		for (int i = 0; i < nDim; ++i)
			u[i] = a[i + k * nDim];
		for (int j = k - 1; j >= 0; --j)
			for (int i = 0; i < nDim; ++i)
				u[i] -= R[j + k * nDim] * Q[i + j * nDim];
		l2 = l2norm(u, nDim);
		for (int i = 0; i < nDim; ++i)
			Q[i + k * nDim] = u[i] / l2;
		for (int j = k; j < nDim; ++j)
		{
			for (int i = 0; i < nDim; ++i)
			{
				u[i] = a[i + j * nDim];
				v[i] = Q[i + k * nDim];
			}
			R[k + j * nDim] = dotprod(u, v, nDim);
		}
	}

	delete[] u;
	delete[] v;
}

__device__
void root_find(
	double *polynomial,
	double *root,
	int nDim_in,
	double tolerance,
	int upperbound)
{
	int nDim = nDim_in - 1;
	double *a = new double[nDim * nDim];
	double *Q = new double[nDim * nDim];
	double *R = new double[nDim * nDim];
	int nTol = 0;
	for (int i = 0; i < nDim; ++i)
		root[i] = 0;

	make_comp_mat(polynomial, a, nDim);

	for (int k = 0; k < upperbound; ++k)
	{
		gram_schmidt(a, Q, R, nDim);
		a = matmult(R, Q, nDim);
		nTol = 0;
		for (int j = 0; j < nDim; ++j)
			for (int i = 0; i < nDim; ++i) {
				if (i > j && fabs(a[i + j * nDim]) > tolerance) ++nTol; }
		if (nTol == 0) break;
	}

	select_diag(root, a, nDim);

	delete[] a;
	delete[] Q;
	delete[] R;
}

__global__
void QRDRoot(
	double *polynomial_image,
	double *root_image,
	double const tolerance,
	int const upperbound,
	int const nDim_image,
	int const nDim_matrix)
{
	int i_image = blockDim.x * blockIdx.x + threadIdx.x;
	if (i_image > nDim_image * nDim_image) return;

	double *polynomial = new double[(nDim_matrix) * (nDim_matrix)];
	double *root = new double [(nDim_matrix - 1) * (nDim_matrix - 1)];

	pixel_mat_select_1d(polynomial_image, polynomial, nDim_matrix, i_image);
	root_find(polynomial, root, nDim_matrix, tolerance, upperbound);
	pixel_mat_write_1d(root_image, root, nDim_matrix - 1, i_image);

	delete[] polynomial;
	delete[] root;
}

/*
__global__
void gram_schmidt(
	double *a_image, 
	double *Q_image, 
	double *R_image,
	int const nDim_image, 
	int const nDim_matrix)
{
	// Assign image pixels to blocks and threads
	int i_image = blockDim.x * blockIdx.x + threadIdx.x;
	if (i_image > nDim_image * nDim_image) return;

	double *a = new double[nDim_matrix * nDim_matrix];
	double *Q = new double[nDim_matrix * nDim_matrix];
	double *R = new double[nDim_matrix * nDim_matrix];
	double *u = new double[nDim_matrix];
	double *v = new double[nDim_matrix];
	double l2 = 0;
	for (int i = 0; i < nDim_matrix * nDim_matrix; ++i)
		a[i] = Q[i] = R[i] = 0;
	for (int i = 0; i < nDim_matrix; ++i)
		u[i] = v[i] = 0;
	
	pixel_mat_select_2d(a_image, a, nDim_matrix, i_image);

	for (int k = 0; k < nDim_matrix; ++k)
	{
		for (int i = 0; i < nDim_matrix; ++i)
			u[i] = a[i + k * nDim_matrix];
		for (int j = k - 1; j >= 0; --j)
			for (int i = 0; i < nDim_matrix; ++i)
				u[i] -= R[j + k * nDim_matrix] * Q[i + j * nDim_matrix];
		l2 = l2norm(u, nDim_matrix);
		for (int i = 0; i < nDim_matrix; ++i)
			Q[i + k * nDim_matrix] = u[i] / l2;
		for (int j = k; j < nDim_matrix; ++j)
		{
			for (int i = 0; i < nDim_matrix; ++i)
			{
				u[i] = a[i + j * nDim_matrix];
				v[i] = Q[i + k * nDim_matrix];
			}
			R[k + j * nDim_matrix] = dotprod(u, v, nDim_matrix);
		}
	}

	delete[] u;
	delete[] v;

	pixel_mat_write_2d(Q_image, R_image, Q, R, nDim_matrix, i_image);
}
*/

